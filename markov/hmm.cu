
#include <hip/hip_runtime.h>
#include <stdio.h>

struct model {
	int states;
	int emissions;
	float* transition;
	float* emission;
	float* initial;
};

#define trans(from,to) (transition[from*states+to])
#define emis(state,obs) (emission[state*states+obs])
#define init(state) (initial[state])

__device__ float par_sum(int state, float *shared, int states) {
	printf("called with state %d of %d\n",state,states);
	int step=1;
	int depth=2;
	do {
		printf("Depth %d Thread %d temp %f states %d\n", depth, state, shared[state], states);
		if (state%depth == 0) {
			printf("Checked Depth %d Thread %d temp %f states %d\n", depth, state, shared[state], states);
			if (state+step < states) {
				// regular case
				// to self add depth far right sibiling
				printf("Summing %f += %f\n",shared[state],shared[state+step]);
				shared[state] += shared[state+step];
			} else {
				printf("Else clause state %d + step %d < states %d\n",state,step,states);
				// loose end case
				// do nothing
				// same as copy from previous
			}
		}
		__syncthreads();
		step = depth;
	} while ((depth*=2) < states);

	printf("Parralel sum %f\n",shared[0]);
	return shared[0];
}
/*
  Takes model and observation sequence of length len and produces
  alfa part of forward-backward algorithm. Normalized for
  numerical stability.
  Assumes alpha is pointer to float array of size m->states * len
  */
// call withc alpha_norm<<grid,block,len*sizeof(int)>>
__global__ void alpha_norm(float* alpha, int states, int emissions, int len, float *transition, float* emission, float *initial, int* obs) {
	int state = blockIdx.x * blockDim.x + threadIdx.x;

	extern __shared__ int s[];
	float *shared = (float *)s;

	if (state >= states) {
		return;
	}

#if 0
	// clear output memory
	for (int i = 0; i < len; i++) {
		alpha[state*len + i] = 0;
	}
#endif

	if (obs[0] > emissions) {
		printf("Observation %d outside model\n",obs[0]);
	}

	printf("Before [%f, %f]\n",alpha[0],alpha[1]);
	// initialize edge case
	shared[state] = init(state) * emis(state,obs[0]);

	// normalize
	alpha[state] = shared[state];
	__syncthreads();
	float sum = par_sum(state, shared, states);
	printf("Sum %f\n",sum);
	alpha[state] /= sum;
	__syncthreads();
	printf("After [%f, %f]\n",alpha[0],alpha[1]);

	printf("initial %d state \n", state);

	// j - observations
	// for each observation sta
	for (int j = 1; j < len - 1; j++) {

		int idx = j * states + state;
		printf("Internal state %d, j %d, idx %d\n",state,j,idx);

		// i - previous state
		// sum for each previous state * transition from previous to current
		sum = 0.f;
		for (int i = 0; i < states; i++) {
			printf("Internal state %d, j %d, i%d\n",state,j,i);
			sum += alpha[states * (j-1) + i] * trans(i,state);
		}

		// normalize
		shared[state]= sum * emis(state,obs[j]);
		alpha[idx] = shared[state];
		__syncthreads();
		sum = par_sum(state, shared, states);
		alpha[idx] /= sum;
		__syncthreads();
	}
	printf("end %d state \n", state);

	__syncthreads();
}

__global__ void beta_norm(float* beta, int states, int emissions, int len, float *transition, float* emission, float *initial, int* obs) {
	int state = blockIdx.x * blockDim.x + threadIdx.x;

	extern __shared__ int s[];
	float *shared = (float *)s;

	if (state >= states) {
		return;
	}

#if 1
	// clear output memory
	for (int i = 0; i < len; i++) {
		beta[state*len + i] = 0;
	}
#endif

	// edge case
	int idx = (len-1) * states + state;
	beta[idx] = 1.f/states;

	// j - observation
	// for each observation from the end
	for (int j = len - 2; j >= 0; j--) {
		int nidx = (j+1) * states + state;
		float sum = 0.f;
		for (int i = 0; i < states; i++) {
			sum += trans(state,i) * emis(state,obs[j+1]) * beta[nidx];
		}
		idx = j * states + state;
		beta[idx] = sum;
		__syncthreads();

		/* shared[state] = beta[idx]; */
		/* __syncthreads(); */
		/* sum = par_sum(state, shared, states); */
		/* beta[idx] /= sum; */

		__syncthreads();
	}

	__syncthreads();
}

